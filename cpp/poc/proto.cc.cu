#include "hip/hip_runtime.h"
// by test

#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <chrono>
#include <iostream>
#include <random>
#include <thread>
#include <unordered_map>
#include <unordered_set>

using std::begin;
using std::cerr;
using std::copy;
using std::cout;
using std::endl;
using std::generate;
using std::max;
using std::min;

using namespace std::chrono;

typedef uint64_t K;
typedef uint64_t M;
typedef float V;
typedef int P;

constexpr uint64_t DIM = 64;
constexpr uint64_t INIT_SIZE = 32 * 4 * 1024 * 1024;  // 134,217,728
constexpr uint64_t BUCKETS_SIZE = 128;
constexpr uint64_t CACHE_SIZE = 2;
constexpr uint64_t BUCKETS_NUM = INIT_SIZE / BUCKETS_SIZE;  // 1,048,576
constexpr K EMPTY_KEY = (K)(0xFFFFFFFFFFFFFFFF);
constexpr M MAX_META = (M)(0xFFFFFFFFFFFFFFFF);

#define CUDA_CHECK(call)                                                 \
  if ((call) != hipSuccess) {                                           \
    hipError_t err = hipGetLastError();                                \
    cerr << "CUDA error calling \"" #call "\", code is " << err << endl; \
  }

__inline__ __device__ uint64_t atomicCAS(uint64_t* address, uint64_t compare,
                                         uint64_t val) {
  return (uint64_t)atomicCAS((unsigned long long*)address,
                             (unsigned long long)compare,
                             (unsigned long long)val);
}

__inline__ __device__ uint64_t atomicMax(uint64_t* address, uint64_t val) {
  return (uint64_t)atomicMax((unsigned long long*)address,
                             (unsigned long long)val);
}

__inline__ __device__ uint64_t atomicMin(uint64_t* address, uint64_t val) {
  return (uint64_t)atomicMin((unsigned long long*)address,
                             (unsigned long long)val);
}

__inline__ __device__ uint64_t atomicExch(uint64_t* address, uint64_t val) {
  return (uint64_t)atomicExch((unsigned long long*)address,
                              (unsigned long long)val);
}

__inline__ __device__ int64_t atomicExch(int64_t* address, int64_t val) {
  return (int64_t)atomicExch((unsigned long long*)address,
                             (unsigned long long)val);
}

__inline__ __device__ int64_t atomicAdd(int64_t* address, const int64_t val) {
  return (int64_t)atomicAdd((unsigned long long*)address, val);
}

__inline__ __device__ uint64_t atomicAdd(uint64_t* address,
                                         const uint64_t val) {
  return (uint64_t)atomicAdd((unsigned long long*)address, val);
}

uint64_t getTimestamp() {
  return duration_cast<milliseconds>(system_clock::now().time_since_epoch())
      .count();
}

inline void __cudaCheckError(const char* file, const int line) {
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }

  // More careful checking. However, this will affect performance.
  // Comment away if needed.
  err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file,
            line, hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}
#define CudaCheckError() __cudaCheckError(__FILE__, __LINE__)

template <typename T>
class FlexMemory {
 public:
  FlexMemory(int size) : ptr_(nullptr) {
    if (!ptr_) {
      size_ = size;
      assert(size_ > 0);
      hipMalloc(&ptr_, sizeof(T) * size_);
    }
  }
  ~FlexMemory() {
    if (!ptr_) hipFree(ptr_);
  }
  V* get(size_t size = 0) {
    if (size > size_) {
      hipFree(ptr_);
      size_ = size;
      assert(size_ > 0);
      hipMalloc(&ptr_, sizeof(T) * size_);
    }
    return ptr_;
  }

 private:
  T* ptr_;
  size_t size_;
};

struct __align__(16) Vector {
  V value[DIM];
};

struct __align__(sizeof(M)) Meta {
  M val;
  //   int prev;
  //   int next;
};

struct Bucket {
  K* keys;          // Device memory
  Meta* metas;      // Device memory
  Vector* cache;    // Device memory
  Vector* vectors;  // Pinned host memory
  Vector* slots1;   // Pinned host memory
  Vector* slots2;   // Pinned host memory
  M min_meta;
  int min_pos;
  int size;
};

struct __align__(32) Table {
  Bucket* buckets;
  unsigned int* locks;
};

inline uint64_t Murmur3Hash(const uint64_t& key) {
  uint64_t k = key;
  k ^= k >> 33;
  k *= UINT64_C(0xff51afd7ed558ccd);
  k ^= k >> 33;
  k *= UINT64_C(0xc4ceb9fe1a85ec53);
  k ^= k >> 33;
  return k;
}

void create_table(Table** table) {
  hipMallocManaged((void**)table, sizeof(Table));
  hipMallocManaged((void**)&((*table)->buckets), BUCKETS_NUM * sizeof(Bucket));
  hipMemset(((*table)->buckets), 0, BUCKETS_NUM * sizeof(Bucket));

  hipMalloc((void**)&((*table)->locks), BUCKETS_NUM * sizeof(int));
  hipMemset((*table)->locks, 0, BUCKETS_NUM * sizeof(unsigned int));

  for (int i = 0; i < BUCKETS_NUM; i++) {
    hipMalloc(&((*table)->buckets[i].keys), BUCKETS_SIZE * sizeof(K));
    hipMemset((*table)->buckets[i].keys, 0xFF, BUCKETS_SIZE * sizeof(K));
    hipMalloc(&((*table)->buckets[i].metas), BUCKETS_SIZE * sizeof(M));
    hipMalloc(&((*table)->buckets[i].cache), CACHE_SIZE * sizeof(Vector));
    hipHostMalloc(&((*table)->buckets[i].vectors),
                   BUCKETS_SIZE * sizeof(Vector), hipHostRegisterMapped);
  }
}

void destroy_table(Table** table) {
  for (int i = 0; i < BUCKETS_NUM; i++) {
    hipFree((*table)->buckets[i].keys);
    hipFree((*table)->buckets[i].metas);
    hipFree((*table)->buckets[i].cache);
    hipHostFree((*table)->buckets[i].vectors);
  }
  hipFree((*table)->locks);
  hipFree((*table)->buckets);
  hipFree(*table);
}

__global__ void write(const Vector* __restrict src, Vector** __restrict dst,
                      int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < N) {
    int vec_index = int(tid / DIM);
    int dim_index = tid % DIM;

    if (dst[vec_index] != nullptr) {
      (*(dst[vec_index])).value[dim_index] = src[vec_index].value[dim_index];
    }
  }
}

__global__ void read(Vector** __restrict src, Vector* __restrict dst, int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < N) {
    int vec_index = int(tid / DIM);
    int dim_index = tid % DIM;
    if (src[vec_index] != nullptr) {
      dst[vec_index].value[dim_index] = (*(src[vec_index])).value[dim_index];
    }
  }
}

__inline__ __device__ void refresh_bucket_meta(Bucket* bucket,
                                               const uint64_t buckets_size) {
  M min_val = MAX_META;
  int min_pos = 0;
  for (int i = 0; i < buckets_size; i++) {
    if (bucket->keys[i] == EMPTY_KEY) {
      continue;
    }
    if (bucket->metas[i].val < min_val) {
      min_pos = i;
      min_val = bucket->metas[i].val;
    }
  }
  atomicExch(&(bucket->min_pos), min_pos);
  atomicExch(&(bucket->min_meta), min_val);
}

__global__ void upsert(const Table* __restrict table, const K* __restrict keys,
                       const M* __restrict metas, Vector** __restrict vectors,
                       int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  int key_pos = -1;
  bool found = false;

  if (tid < N) {
    int key_idx = tid;
    int bkt_idx = keys[tid] % BUCKETS_NUM;
    const K insert_key = keys[tid];
    bool release_lock = false;

    while (!release_lock) {
      if (atomicExch(&(table->locks[bkt_idx]), 1u) == 0u) {
        Bucket* bucket = &(table->buckets[bkt_idx]);
        for (int i = 0; i < BUCKETS_SIZE; i++) {
          if (bucket->keys[i] == insert_key) {
            found = true;
            key_pos = i;
            break;
          }
        }
        for (int i = 0; i < BUCKETS_SIZE; i++) {
          K old_key = atomicCAS(&(bucket->keys[i]), EMPTY_KEY, insert_key);
          if (old_key == EMPTY_KEY) {
            key_pos = i;
            break;
          }
        }
        if (metas[key_idx] >= bucket->min_meta || found ||
            bucket->size < BUCKETS_SIZE) {
          if (!found) {
            key_pos = key_pos == -1 ? bucket->min_pos : key_pos;
            atomicAdd(&(bucket->size), 1);
            atomicMin(&(bucket->size), BUCKETS_SIZE);
          }
          atomicExch(&(bucket->keys[key_pos]), insert_key);
          atomicExch(&(bucket->metas[key_pos].val), metas[key_idx]);

          M tmp_min_val = MAX_META;
          int tmp_min_pos = 0;
          for (int i = 0; i < BUCKETS_SIZE; i++) {
            if (bucket->keys[i] == EMPTY_KEY) {
              break;
            }
            if (bucket->metas[i].val < tmp_min_val) {
              tmp_min_pos = i;
              tmp_min_val = bucket->metas[i].val;
            }
          }
          atomicExch(&(bucket->min_pos), tmp_min_pos);
          atomicExch(&(bucket->min_meta), tmp_min_val);
          atomicCAS((uint64_t*)&(vectors[tid]), (uint64_t)(nullptr),
                    (uint64_t)((Vector*)(bucket->vectors) + key_pos));
        }
        release_lock = true;
        atomicExch(&(table->locks[bkt_idx]), 0u);
      }
    }
  }
}

__global__ void upsert_fast(const Table* __restrict table,
                            const K* __restrict keys, const M* __restrict metas,
                            Vector** __restrict vectors,
                            const bool* __restrict d_found,
                            const int* __restrict offset, int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  int key_pos = -1;

  if (tid < N) {
    int key_idx = tid;
    int bkt_idx = keys[tid] % BUCKETS_NUM;
    const K insert_key = keys[tid];
    bool release_lock = false;
    bool found = d_found[key_idx];
    bool unfilled = false;

    while (!release_lock) {
      if (atomicExch(&(table->locks[bkt_idx]), 1u) == 0u) {
        Bucket* bucket = &(table->buckets[bkt_idx]);
        if (found) {
          key_pos = offset[key_idx];
        } else {
          for (int i = 0; i < BUCKETS_SIZE; i++) {
            K old_key = atomicCAS(&(bucket->keys[i]), EMPTY_KEY, insert_key);
            if (old_key == EMPTY_KEY) {
              key_pos = i;
              unfilled = true;
              break;
            }
          }
        }

        if (metas[key_idx] >= bucket->min_meta || found || unfilled) {
          if (!found) {
            key_pos = key_pos == -1 ? bucket->min_pos : key_pos;
          }
          atomicExch(&(bucket->keys[key_pos]), insert_key);
          atomicExch(&(bucket->metas[key_pos].val), metas[key_idx]);

          refresh_bucket_meta(bucket, BUCKETS_SIZE);
          atomicCAS((uint64_t*)&(vectors[tid]), (uint64_t)(nullptr),
                    (uint64_t)((Vector*)(bucket->vectors) + key_pos));
        }
        release_lock = true;
        atomicExch(&(table->locks[bkt_idx]), 0u);
      }
    }
  }
}

constexpr uint64_t NUM_THREADS = 1024;

constexpr uint64_t NUM_THREADS_UPSERT = 256;

__global__ void upsert_cg_sharedmem(const Table* __restrict table,
                                    const K* __restrict keys,
                                    const M* __restrict metas,
                                    Vector** __restrict vectors, int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  int key_pos = -1;
  bool found = false;
  bool empty = false;

  extern __shared__ unsigned char s[];
  K* bucket_keys = (K*)s;
  M* bucket_metas = (M*)s;

  if (tid < N) {
    int key_idx = tid / (BUCKETS_SIZE / 32);
    int bkt_idx = keys[key_idx] % BUCKETS_NUM;
    const K insert_key = keys[key_idx];
    bool release_lock = false;

    int start_pos = ((tid * (BUCKETS_SIZE / 32)) % BUCKETS_SIZE);

    while (!release_lock && start_pos == 0) {
      if (atomicExch(&(table->locks[bkt_idx]), 1u) == 0u) {
        release_lock = true;
      }
    }
    Bucket* bucket = &(table->buckets[bkt_idx]);

    printf("insert 222\n");
// read to shared memory
#pragma unroll
    for (int i = 0; i < BUCKETS_SIZE / 32; i++) {
      bucket_keys[start_pos + i] = bucket->keys[start_pos + i];
    }
    __syncthreads();

// found if existed
#pragma unroll
    for (int i = 0; i < BUCKETS_SIZE / 32; i++) {
      if (bucket_keys[start_pos + i] == insert_key) {
        found = true;
        key_pos = start_pos + i;
        break;
      }
    }

    int sync_found = __any_sync(0xFFFFFFFF, int(found));
    if (!sync_found) {
      // found a empty postion
      if (start_pos == 0) {
#pragma unroll
        for (int i = 0; i < BUCKETS_SIZE; i++) {
          if (bucket_keys[i] == EMPTY_KEY) {
            key_pos = start_pos + i;
            empty = true;
            printf("insert 555, %d\n", int(empty));
            atomicExch(&(bucket->keys[key_pos]), insert_key);
            atomicExch(&(bucket->metas[key_pos].val), metas[key_idx]);
            break;
          }
        }
      }
      printf("insert 444, %d\n", int(empty));
      for (int i = 0; i < BUCKETS_SIZE / 32; i++) {
        bucket_metas[start_pos + i] = bucket->metas[start_pos + i].val;
      }
      __syncthreads();

      if (!empty && start_pos == 0) {
        int min_meta_pos = -1;
        M min_meta = MAX_META;
#pragma unroll
        for (int i = 0; i < BUCKETS_SIZE; i++) {
          if (min_meta >= bucket_metas[i]) {
            min_meta_pos = i;
            min_meta = bucket_metas[i];
          }
        }
        if (metas[key_idx] > min_meta) {
          atomicExch(&(bucket->keys[min_meta_pos]), insert_key);
          atomicExch(&(bucket->metas[min_meta_pos].val), metas[key_idx]);
        }
      }
    }
    //
    //     // the found thread continue
    //     if (found) {
    //       printf("insert 2\n");
    //       atomicExch(&(bucket->keys[key_pos]), insert_key);
    //       atomicExch(&(bucket->metas[key_pos].val), metas[key_idx]);
    //     }
    //
    if (start_pos == 0) {
      atomicExch(&(table->locks[bkt_idx]), 0u);
    }
  }
}

__global__ void upsert_cg(const Table* __restrict table,
                          const K* __restrict keys, const M* __restrict metas,
                          Vector** __restrict vectors, int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  int key_pos = -1;
  bool found = false;
  bool empty = false;
  extern __shared__ unsigned char s[];
  K* bucket_keys = (K*)s;
  M* bucket_metas =
      (M*)(bucket_keys + NUM_THREADS_UPSERT * (BUCKETS_SIZE / 32));

  int i;

  if (tid < N) {
    int key_idx = tid / (BUCKETS_SIZE / 32);
    int bkt_idx = keys[key_idx] % BUCKETS_NUM;
    const K insert_key = keys[key_idx];
    bool release_lock = false;

    int start_pos = ((tid * (BUCKETS_SIZE / 32)) % BUCKETS_SIZE);

    while (!release_lock && start_pos == 0) {
      if (atomicExch(&(table->locks[bkt_idx]), 1u) == 0u) {
        release_lock = true;
      }
    }
    Bucket* bucket = &(table->buckets[bkt_idx]);

// read to shared memory
#pragma unroll
    for (i = 0; i < BUCKETS_SIZE / 32; i++) {
      bucket_keys[start_pos + i] = bucket->keys[start_pos + i];
      bucket_metas[start_pos + i] = bucket->metas[start_pos + i].val;
    }

// found if existed
#pragma unroll
    for (i = 0; i < BUCKETS_SIZE / 32; i++) {
      if (bucket_keys[start_pos + i] == insert_key) {
        found = true;
        key_pos = start_pos + i;
      }
      if (bucket_keys[i] == EMPTY_KEY) {
        key_pos = i;
        empty = true;
      }
    }

    int sync_found = __any_sync(0xFFFFFFFF, int(found));
    if (!sync_found) {
      // found a empty postion
      if (start_pos == 0) {
        atomicExch(&(bucket->keys[key_pos]), insert_key);
        atomicExch(&(bucket->metas[key_pos].val), metas[key_idx]);
      }

      if (!empty && start_pos == 0) {
        //         int min_meta_pos = -1;
        key_pos = -1;
        M min_meta = MAX_META;
#pragma unroll
        for (i = 0; i < BUCKETS_SIZE; i++) {
          if (min_meta >= bucket_metas[i]) {
            key_pos = i;
            min_meta = bucket_metas[i];
          }
        }
        if (metas[key_idx] > min_meta) {
          atomicExch(&(bucket->keys[key_pos]), insert_key);
          atomicExch(&(bucket->metas[key_pos].val), metas[key_idx]);
        }
      }
    }

    // the found thread continue
    if (found) {
      atomicExch(&(bucket->keys[key_pos]), insert_key);
      atomicExch(&(bucket->metas[key_pos].val), metas[key_idx]);
    }

    if (start_pos == 0) {
      atomicExch(&(table->locks[bkt_idx]), 0u);
    }
  }
}

__global__ void lookup(const Table* __restrict table, const K* __restrict keys,
                       Vector** __restrict vectors, bool* __restrict found,
                       int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) {
    int key_idx = tid / BUCKETS_SIZE;
    int key_pos = tid % BUCKETS_SIZE;
    int bkt_idx = keys[key_idx] % BUCKETS_NUM;
    K target_key = keys[key_idx];
    Bucket* bucket = &(table->buckets[bkt_idx]);

    if (bucket->keys[key_pos] == target_key) {
      vectors[key_idx] = (Vector*)&(bucket->vectors[key_pos]);
      found[key_idx] = true;
    }
  }
}

__global__ void lookup_fast(const Table* __restrict table,
                            const K* __restrict keys,
                            Vector** __restrict vectors, bool* __restrict found,
                            int N) {
  extern __shared__ unsigned char s[];
  int* possible_key_pos = (int*)s;
  //   int possible_key_pos[4];
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) {
    int key_idx = tid / BUCKETS_SIZE;
    int key_pos = tid % BUCKETS_SIZE;
    int bkt_idx = keys[key_idx] % BUCKETS_NUM;
    K target_key = keys[key_idx];
    Bucket* bucket = &(table->buckets[bkt_idx]);

    //     unsigned int pre =
    //         __ballot_sync(0xFFFFFFFF, int(bucket->keys[key_pos] ==
    //         target_key));
    //       __syncthreads();
    possible_key_pos[tid % BUCKETS_SIZE] =
        int(bucket->keys[key_pos] == target_key);
    int max_pos = -1;
    if (tid % BUCKETS_SIZE == 0) {
      for (int i = 0; i < BUCKETS_SIZE; i++) {
        if (possible_key_pos[i] > 0) {
          vectors[key_idx] = (Vector*)&(bucket->vectors[i]);
          found[key_idx] = true;
          break;
        }
      }
    }
  }
}

__global__ void lookup(const Table* __restrict table, const K* __restrict keys,
                       M* __restrict metas, Vector** __restrict vectors,
                       bool* __restrict found, int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) {
    int key_idx = tid / BUCKETS_SIZE;
    int key_pos = tid % BUCKETS_SIZE;
    int bkt_idx = keys[key_idx] % BUCKETS_NUM;
    K target_key = keys[key_idx];
    Bucket* bucket = &(table->buckets[bkt_idx]);

    if (bucket->keys[key_pos] == target_key) {
      metas[key_idx] = bucket->metas[key_pos].val;
      vectors[key_idx] = (Vector*)&(bucket->vectors[key_pos]);
      found[key_idx] = true;
    }
  }
}

__global__ void lookup_for_upsert(const Table* __restrict table,
                                  const K* __restrict keys,
                                  bool* __restrict found,
                                  int* __restrict offset, int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) {
    int key_idx = tid / BUCKETS_SIZE;
    int key_pos = tid % BUCKETS_SIZE;
    int bkt_idx = keys[key_idx] % BUCKETS_NUM;
    K target_key = keys[key_idx];
    Bucket* bucket = &(table->buckets[bkt_idx]);

    if (bucket->keys[key_pos] == target_key) {
      found[key_idx] = true;
      atomicExch((int*)&(offset[key_idx]), key_pos);
    }
  }
}

__global__ void remove(const Table* __restrict table, const K* __restrict keys,
                       int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) {
    int key_idx = tid / BUCKETS_SIZE;
    int key_pos = tid % BUCKETS_SIZE;
    int bkt_idx = keys[key_idx] % BUCKETS_NUM;
    K target_key = keys[key_idx];
    Bucket* bucket = &(table->buckets[bkt_idx]);

    K old_key = atomicCAS((K*)&bucket->keys[key_pos], target_key, EMPTY_KEY);
    if (old_key == target_key) {
      atomicExch((K*)&(bucket->metas[key_pos].val), MAX_META);
      atomicDec((unsigned int*)&(bucket->size), BUCKETS_SIZE);
    }
  }
}

__global__ void clear(Table* table, int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) {
    int key_idx = tid % BUCKETS_SIZE;
    int bkt_idx = tid / BUCKETS_SIZE;
    Bucket* bucket = &(table->buckets[bkt_idx]);
    atomicExch((K*)&(bucket->keys[key_idx]), EMPTY_KEY);
    atomicExch((K*)&(bucket->metas[key_idx].val), MAX_META);
    if (key_idx == 0) atomicExch(&(bucket->size), 0);
  }
}

__global__ void size(Table* table, size_t* size, int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < N) {
    for (int i = 0; i < BUCKETS_SIZE; i++) {
      if (table->buckets[tid].keys[i] != EMPTY_KEY) {
        atomicAdd((unsigned long long int*)&(size[tid]), 1);
      }
    }
  }
}

__global__ void size_new(Table* table, size_t* size, int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) {
    atomicAdd((unsigned long long int*)(size), table->buckets[tid].size);
  }
}

__global__ void dump(const Table* table, K* d_key, Vector* d_val,
                     const size_t offset, const size_t search_length,
                     size_t* d_dump_counter) {
  extern __shared__ unsigned char s[];
  K* smem = (K*)s;
  K* block_result_key = smem;
  Vector* block_result_val = (Vector*)&(smem[blockDim.x]);
  __shared__ size_t block_acc;
  __shared__ size_t global_acc;

  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (threadIdx.x == 0) {
    block_acc = 0;
  }
  __syncthreads();

  if (tid < search_length) {
    int bkt_idx = (tid + offset) / BUCKETS_SIZE;
    int key_idx = (tid + offset) % BUCKETS_SIZE;
    Bucket* bucket = &(table->buckets[bkt_idx]);

    if (bucket->keys[key_idx] != EMPTY_KEY) {
      size_t local_index = atomicAdd(&block_acc, 1);
      block_result_key[local_index] = bucket->keys[key_idx];
      for (int i = 0; i < DIM; i++) {
        block_result_val[local_index].value[i] =
            bucket->vectors[key_idx].value[i];
      }
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    //     printf("block_acc=%llu \n", block_acc);
    global_acc = atomicAdd(d_dump_counter, block_acc);
  }
  __syncthreads();

  if (threadIdx.x < block_acc) {
    d_key[global_acc + threadIdx.x] = block_result_key[threadIdx.x];
    for (int i = 0; i < DIM; i++) {
      d_val[global_acc + threadIdx.x].value[i] =
          block_result_val[threadIdx.x].value[i];
    }
  }
}

template <typename T>
void create_random_keys_test(T* h_keys, M* h_metas, int KEY_NUM) {
  std::unordered_set<T> numbers;
  std::random_device rd;
  std::mt19937_64 eng(rd());
  std::uniform_int_distribution<T> distr;
  T max_key = 0;
  T min_key = 0xFFFFFFFFFFFFFFFF;
  int i = 0;

  while (numbers.size() < KEY_NUM) {
    T tmp = distr(eng);
    if (Murmur3Hash(tmp) % BUCKETS_NUM == 0) numbers.insert(tmp);
  }
  for (const T num : numbers) {
    h_keys[i] = Murmur3Hash(num);
    max_key = max_key < h_keys[i] ? h_keys[i] : max_key;
    min_key = min_key > h_keys[i] ? h_keys[i] : min_key;
    h_metas[i] = getTimestamp() + i;
    i++;
  }
  std::cout << "create_random_keys: " << max_key << " " << min_key << std::endl;
}

std::unordered_set<K> numbers;
template <typename T>
void create_random_keys(T* h_keys, M* h_metas, int KEY_NUM) {
  std::random_device rd;
  std::mt19937_64 eng(rd());
  std::uniform_int_distribution<T> distr;
  T max_key = 0;
  T min_key = 0xFFFFFFFFFFFFFFFF;
  int i = 0;

  while (numbers.size() < KEY_NUM) {
    T tmp = distr(eng);
    numbers.insert(Murmur3Hash(tmp));
  }
  for (const T num : numbers) {
    h_keys[i] = num;
    max_key = max_key < h_keys[i] ? h_keys[i] : max_key;
    min_key = min_key > h_keys[i] ? h_keys[i] : min_key;
    h_metas[i] = (M)(h_keys[i]);
    i++;
  }
  //   std::cout << "create_random_keys: " << max_key << " " << min_key <<
  //   std::endl;
}

int main() {
  constexpr uint64_t KEY_NUM = 1 * 1024 * 1024;
  constexpr uint64_t TEST_TIMES = 1;

  int total_size = 0;
  size_t max_bucket_len = 0;
  size_t min_bucket_len = KEY_NUM;
  int found_num = 0;
  std::unordered_map<int, int> size2length;

  K* h_keys;
  K* h_dump_keys;
  M* h_metas;
  Vector* h_vectors;
  Vector* h_dump_vectors;
  size_t* h_size;
  size_t h_counter;
  bool* h_found;

  hipHostMalloc(&h_keys, KEY_NUM * sizeof(K));               // 8MB
  hipHostMalloc(&h_dump_keys, KEY_NUM * sizeof(K));          // 8MB
  hipHostMalloc(&h_metas, KEY_NUM * sizeof(M));              // 8MB
  hipHostMalloc(&h_vectors, KEY_NUM * sizeof(Vector));       // 256MB
  hipHostMalloc(&h_dump_vectors, KEY_NUM * sizeof(Vector));  // 256MB
  hipHostMalloc(&h_size, BUCKETS_NUM * sizeof(size_t));      // 8MB
  hipHostMalloc(&h_found, KEY_NUM * sizeof(bool));           // 4MB

  hipMemset(h_vectors, 0, KEY_NUM * sizeof(Vector));

  create_random_keys<K>(h_keys, h_metas, KEY_NUM);

  Table* d_table;
  K* d_keys;
  K* d_dump_keys;
  M* d_metas = nullptr;
  Vector* d_vectors;
  Vector* d_dump_vectors;
  Vector** d_vectors_ptr;
  size_t* d_size;
  size_t* d_counter;
  bool* d_found;
  int* d_offset;

  hipMalloc(&d_dump_keys, KEY_NUM * sizeof(K));          // 8MB
  hipMalloc(&d_keys, KEY_NUM * sizeof(K));               // 8MB
  hipMalloc(&d_metas, KEY_NUM * sizeof(M));              // 8MB
  hipMalloc(&d_vectors, KEY_NUM * sizeof(Vector));       // 256MB
  hipMalloc(&d_dump_vectors, KEY_NUM * sizeof(Vector));  // 256MB
  hipMalloc(&d_vectors_ptr, KEY_NUM * sizeof(Vector*));  // 8MB
  hipMalloc(&d_size, BUCKETS_NUM * sizeof(size_t));      // 8MB
  hipMalloc(&d_found, KEY_NUM * sizeof(bool));           // 4MB
  hipMalloc(&d_offset, KEY_NUM * sizeof(int));           // 4MB
  hipMalloc(&d_counter, sizeof(size_t));                 // 4MB

  hipMemcpy(d_keys, h_keys, KEY_NUM * sizeof(K), hipMemcpyHostToDevice);
  hipMemcpy(d_metas, h_metas, KEY_NUM * sizeof(M), hipMemcpyHostToDevice);

  hipMemset(d_vectors, 1, KEY_NUM * sizeof(Vector));
  hipMemset(d_dump_vectors, 0, KEY_NUM * sizeof(Vector));
  hipMemset(d_vectors_ptr, 0, KEY_NUM * sizeof(Vector*));
  hipMemset(d_found, 0, KEY_NUM * sizeof(bool));
  hipMemset(d_offset, 0, KEY_NUM * sizeof(int));
  hipMemset(d_counter, 0, sizeof(size_t));
  hipMemset(d_size, 0, BUCKETS_NUM * sizeof(size_t));

  hipStream_t stream;
  hipStreamCreate(&stream);

  uint64_t N = KEY_NUM;
  uint64_t NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

  create_table(&d_table);

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  size_t shared_mem_size = deviceProp.sharedMemPerBlock;

  for (int i = 0; i < TEST_TIMES; i++) {
    found_num = 0;
    // upsert test

    // lookup test
    N = BUCKETS_SIZE * KEY_NUM;
    NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    hipMemset(d_found, 0, KEY_NUM * sizeof(bool));
    hipDeviceSynchronize();

    auto start_lookup_for_upsert = std::chrono::steady_clock::now();
    lookup_for_upsert<<<NUM_BLOCKS, NUM_THREADS>>>(d_table, d_keys, d_found,
                                                   d_offset, N);
    hipDeviceSynchronize();
    auto end_lookup_for_upsert = std::chrono::steady_clock::now();
    std::chrono::duration<double> diff_lookup_for_upsert =
        end_lookup_for_upsert - start_lookup_for_upsert;

    N = KEY_NUM;
    NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

    auto start_upsert = std::chrono::steady_clock::now();
    upsert_fast<<<NUM_BLOCKS, NUM_THREADS>>>(
        d_table, d_keys, d_metas, d_vectors_ptr, d_found, d_offset, N);

    //     N = KEY_NUM * 32;
    //     NUM_BLOCKS = (N + NUM_THREADS_UPSERT - 1) / NUM_THREADS_UPSERT;
    //     int upsert_cg_shared_mem_size =
    //         (sizeof(K) + sizeof(M)) * NUM_THREADS_UPSERT * (BUCKETS_SIZE /
    //         32);
    //     std::cout << "shared mem=" << shared_mem_size << ", "
    //               << upsert_cg_shared_mem_size << std::endl;
    //     std::cout << "num blocks & num threads=" << NUM_BLOCKS << ", "
    //               << NUM_THREADS_UPSERT << std::endl;
    //
    //     upsert_cg<<<NUM_BLOCKS, NUM_THREADS_UPSERT,
    //     upsert_cg_shared_mem_size, 0>>>(
    //         d_table, d_keys, d_metas, d_vectors_ptr, N);
    //     CudaCheckError();
    hipDeviceSynchronize();
    auto end_upsert = std::chrono::steady_clock::now();
    std::chrono::duration<double> diff_upsert = end_upsert - start_upsert;

    N = KEY_NUM * DIM;
    NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

    auto start_write = std::chrono::steady_clock::now();
    write<<<NUM_BLOCKS, NUM_THREADS>>>(d_vectors, d_vectors_ptr, N);
    hipDeviceSynchronize();
    auto end_write = std::chrono::steady_clock::now();
    std::chrono::duration<double> diff_write = end_write - start_write;

    //     // size test
    //     N = BUCKETS_NUM;
    //     NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    //     size<<<NUM_BLOCKS, NUM_THREADS>>>(d_table, d_size, N);
    //     hipDeviceSynchronize();

    // size test
    hipMemset(d_size, 0, BUCKETS_NUM * sizeof(size_t));
    N = BUCKETS_NUM;
    NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    size_new<<<NUM_BLOCKS, NUM_THREADS>>>(d_table, d_size, N);
    hipDeviceSynchronize();
    hipMemcpy(h_size, d_size, BUCKETS_NUM * sizeof(size_t),
               hipMemcpyDeviceToHost);
    cout << "after upsert, size=" << h_size[0] << endl;

    // dump:
    hipMemset(d_counter, 0, sizeof(d_counter));
    hipMemset(d_vectors, 0, KEY_NUM * sizeof(Vector));

    std::cout << "shared_mem_size=" << shared_mem_size << std::endl;

    size_t search_length = INIT_SIZE;
    size_t block_size = shared_mem_size * 0.5 / (sizeof(K) + sizeof(Vector));
    cout << "dump block_Size=" << block_size << endl;
    block_size = block_size <= 1024 ? block_size : 1024;
    assert(block_size > 0 &&
           "nv::merlinhash: block_size <= 0, the KV size may be too large!");
    size_t shared_size = sizeof(K) * block_size + sizeof(Vector) * block_size;
    const int grid_size = (search_length - 1) / (block_size) + 1;

    dump<<<grid_size, block_size, shared_size, stream>>>(
        d_table, d_dump_keys, d_dump_vectors, 0, search_length, d_counter);
    hipDeviceSynchronize();
    hipMemcpy(&h_counter, d_counter, sizeof(d_counter),
               hipMemcpyDeviceToHost);
    cout << "dump, h_counter=" << h_counter << endl;

    hipMemcpy(h_dump_keys, d_dump_keys, KEY_NUM * sizeof(K),
               hipMemcpyDeviceToHost);
    hipMemcpy(h_dump_vectors, d_dump_vectors, KEY_NUM * sizeof(Vector),
               hipMemcpyDeviceToHost);
    int error_dump_count = 0;
    for (int i = 0; i < KEY_NUM; i++) {
      for (int j = 0; j < DIM; j++) {
        V tmp = h_dump_vectors[i].value[j];
        int* tmp_int = reinterpret_cast<int*>((V*)(&tmp));
        if (*tmp_int != 16843009) error_dump_count++;
      }
    }
    cout << "check1: dump error_count=" << error_dump_count << endl;
    error_dump_count = 0;
    for (int i = 0; i < KEY_NUM; i++) {
      if (numbers.end() == numbers.find(h_dump_keys[i])) error_dump_count++;
    }
    cout << "check2: dump error_count=" << error_dump_count << endl;

    // lookup test
    N = BUCKETS_SIZE * KEY_NUM;
    NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    hipMemset(d_vectors_ptr, 0, KEY_NUM * sizeof(Vector*));
    hipMemset(d_found, 0, KEY_NUM * sizeof(bool));
    hipMemset(d_metas, 0, KEY_NUM * sizeof(M));
    hipDeviceSynchronize();

    auto start_lookup = std::chrono::steady_clock::now();
    hipDeviceSynchronize();
    lookup<<<NUM_BLOCKS, NUM_THREADS>>>(d_table, d_keys, d_metas, d_vectors_ptr,
                                        d_found, N);
    //     lookup_fast<<<NUM_BLOCKS, NUM_THREADS, 4 * NUM_THREADS, 0>>>(
    //         d_table, d_keys, d_vectors_ptr, d_found, N);
    hipDeviceSynchronize();
    auto end_lookup = std::chrono::steady_clock::now();
    std::chrono::duration<double> diff_lookup = end_lookup - start_lookup;

    N = KEY_NUM * DIM;
    NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    hipMemset(d_vectors, 0, KEY_NUM * sizeof(Vector));
    auto start_read = std::chrono::steady_clock::now();
    read<<<NUM_BLOCKS, NUM_THREADS>>>(d_vectors_ptr, d_vectors, N);
    hipDeviceSynchronize();
    auto end_read = std::chrono::steady_clock::now();
    std::chrono::duration<double> diff_read = end_read - start_read;

    // remove:
    hipMemset(d_size, 0, BUCKETS_NUM * sizeof(size_t));
    int remove_key_num = KEY_NUM / 2;
    N = remove_key_num * BUCKETS_SIZE;
    NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    remove<<<NUM_BLOCKS, NUM_THREADS>>>(d_table, d_keys, N);
    hipDeviceSynchronize();
    N = BUCKETS_NUM;
    NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    size_new<<<NUM_BLOCKS, NUM_THREADS>>>(d_table, d_size, N);
    hipDeviceSynchronize();
    hipMemcpy(h_size, d_size, BUCKETS_NUM * sizeof(size_t),
               hipMemcpyDeviceToHost);
    cout << "after remove, size=" << h_size[0] << endl;

    // clear:
    hipMemset(d_size, 0, BUCKETS_NUM * sizeof(size_t));
    N = BUCKETS_NUM * BUCKETS_SIZE;
    NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    clear<<<NUM_BLOCKS, NUM_THREADS>>>(d_table, N);
    hipDeviceSynchronize();

    N = BUCKETS_NUM;
    NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    size_new<<<NUM_BLOCKS, NUM_THREADS>>>(d_table, d_size, N);
    hipDeviceSynchronize();
    hipMemcpy(h_size, d_size, BUCKETS_NUM * sizeof(size_t),
               hipMemcpyDeviceToHost);
    cout << "after clear, size=" << h_size[0] << endl;

    printf("[timing] upsert=%.2fms, lookup_for_upsert=%.2fms, write=%.2fms\n",
           diff_upsert.count() * 1000, diff_lookup_for_upsert.count() * 1000,
           diff_write.count() * 1000);
    printf("[timing] lookup=%.2fms, read = % .2fms\n ",
           diff_lookup.count() * 1000, diff_read.count() * 1000);
    hipMemcpy(h_size, d_size, BUCKETS_NUM * sizeof(size_t),
               hipMemcpyDeviceToHost);

    hipMemcpy(h_found, d_found, KEY_NUM * sizeof(bool),
               hipMemcpyDeviceToHost);
    hipMemcpy(h_vectors, d_vectors, KEY_NUM * sizeof(Vector),
               hipMemcpyDeviceToHost);
    hipMemcpy(&h_counter, d_counter, sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_metas, d_metas, KEY_NUM * sizeof(M), hipMemcpyDeviceToHost);
    for (int i = 0; i < KEY_NUM; i++) {
      if (h_found[i]) found_num++;
    }

    for (int i = 0; i < BUCKETS_NUM; i++) {
      total_size += h_size[i];
      if (size2length.find(h_size[i]) != size2length.end()) {
        size2length[h_size[i]] += 1;
      } else {
        size2length[h_size[i]] = 1;
      }
      max_bucket_len = max(max_bucket_len, h_size[i]);
      min_bucket_len = min(min_bucket_len, h_size[i]);
    }

    cout << "Capacity = " << INIT_SIZE << ", total_size = " << total_size
         << ", h_size[0] = " << h_size[0] << ", h_counter = " << h_counter
         << ", max_bucket_len = " << max_bucket_len
         << ", min_bucket_len = " << min_bucket_len
         << ", found_num = " << found_num << endl;
    //     assert(found_num == 128);
  }
  destroy_table(&d_table);
  hipStreamDestroy(stream);

  int error_count = 0;
  for (int i = 0; i < KEY_NUM; i++) {
    for (int j = 0; j < DIM; j++) {
      V tmp = h_vectors[i].value[j];
      int* tmp_int = reinterpret_cast<int*>((V*)(&tmp));
      if (*tmp_int != 16843009) error_count++;
    }
  }
  cout << "check1: error_count=" << error_count << endl;

  error_count = 0;
  for (int i = 0; i < KEY_NUM; i++) {
    if (h_keys[i] != h_metas[i]) error_count++;
  }
  cout << "check2:error_count=" << error_count << endl;

  uint64_t min_meta = 0xFFFFFFFFFFFFFFFF;
  for (int i = 0; i < KEY_NUM; i++) {
    if (!h_found[i]) continue;
    min_meta = h_metas[i] < min_meta ? h_metas[i] : min_meta;
  }
  int bigger = 0;
  int smaller = 0;
  for (int i = 0; i < KEY_NUM; i++) {
    if (h_keys[i] > min_meta) bigger++;
    if (h_keys[i] < min_meta) smaller++;
  }
  cout << "check3:bigger=" << bigger << endl;
  cout << "check3:smaller=" << smaller << endl;
  //   for(auto n: size2length){
  //     cout << n.first << "    " << n.second << endl;
  //   }

  hipHostFree(h_keys);
  hipHostFree(h_dump_keys);
  hipHostFree(h_vectors);
  hipHostFree(h_dump_vectors);
  hipHostFree(h_metas);
  hipHostFree(h_size);
  hipHostFree(h_found);

  hipFree(d_keys);
  hipFree(d_dump_keys);
  hipFree(d_metas);
  hipFree(d_vectors);
  hipFree(d_dump_vectors);
  hipFree(d_vectors_ptr);
  hipFree(d_size);
  hipFree(d_found);
  hipFree(d_offset);
  hipFree(d_counter);

  cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}
