#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;
namespace cg = cooperative_groups;

#include <algorithm>
#include <chrono>
#include <iostream>
#include <random>
#include <thread>
#include <unordered_set>

typedef uint64_t K;
typedef uint64_t M;
typedef float V;

template <class K>
void create_random_keys(K *h_keys, int KEY_NUM) {
  std::unordered_set<K> numbers;
  std::random_device rd;
  std::mt19937_64 eng(rd());
  std::uniform_int_distribution<K> distr;
  int i = 0;

  while (numbers.size() < KEY_NUM) {
    numbers.insert(distr(eng));
  }
  for (const K num : numbers) {
    h_keys[i] = num;
    i++;
  }
}

template <class K>
void create_continuous_keys(K *h_keys, int KEY_NUM, K start = 0) {
  for (K i = 0; i < KEY_NUM; i++) {
    h_keys[i] = start + static_cast<K>(i);
  }
}

template <class M>
struct Meta {
  M val;
};

constexpr uint64_t EMPTY_KEY = std::numeric_limits<uint64_t>::max();
constexpr uint64_t MAX_META = std::numeric_limits<uint64_t>::max();
constexpr uint64_t EMPTY_META = std::numeric_limits<uint64_t>::min();

template <class K>
struct Bucket {
  K *keys;         // HBM
  Meta<M> *metas;  // HBM
  V *cache;        // HBM(optional)
  V *vectors;      // Pinned memory or HBM

  /* For upsert_kernel without user specified metas
     recording the current meta, the cur_meta will
     increment by 1 when a new inserting happens. */
  M cur_meta;

  /* min_meta and min_pos is for or upsert_kernel
     with user specified meta. They record the minimum
     meta and its pos in the bucket. */
  M min_meta;
  int min_pos;
};

constexpr int KEY_NUM = 1024 * 1024;
constexpr int INIT_SIZE = KEY_NUM * 64;
constexpr int MAX_BUCKET_SIZE = 128;
constexpr const size_t BLOCK_SIZE = 128;
constexpr int TILE_SIZE = 8;
constexpr const size_t N = KEY_NUM * TILE_SIZE;
constexpr const size_t GRID_SIZE = ((N)-1) / BLOCK_SIZE + 1;
constexpr int BUCKETS_NUM = INIT_SIZE / MAX_BUCKET_SIZE;

__inline__ __device__ uint64_t Murmur3HashDevice(uint64_t const &key) {
  uint64_t k = key;
  k ^= k >> 33;
  k *= UINT64_C(0xff51afd7ed558ccd);
  k ^= k >> 33;
  k *= UINT64_C(0xc4ceb9fe1a85ec53);
  k ^= k >> 33;
  return k;
}

template <class Key>
__global__ void upsert_kernel(const Key *__restrict keys,
                              const Bucket<K> *__restrict buckets,
                              int *__restrict d_sizes, V **__restrict vectors,
                              int *__restrict src_offset, size_t N) {
  size_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  auto g = cg::tiled_partition<TILE_SIZE>(cg::this_thread_block());
  int rank = g.thread_rank();

  for (size_t t = tid; t < N; t += blockDim.x * gridDim.x) {

    int key_pos = -1;
    bool found_or_empty = false;
    size_t key_idx = t / TILE_SIZE;
    Key insert_key = *(keys + key_idx);
//    Key hashed_key = Murmur3HashDevice(insert_key);
    size_t bkt_idx = insert_key & (BUCKETS_NUM - 1);
    size_t start_idx = insert_key & (MAX_BUCKET_SIZE - 1);
    int src_lane;

    const Bucket<Key> *bucket = buckets + bkt_idx;

#pragma unroll
    for (uint32_t tile_offset = 0; tile_offset < MAX_BUCKET_SIZE;
         tile_offset += TILE_SIZE) {
      size_t key_offset = (start_idx + tile_offset + rank) & (MAX_BUCKET_SIZE - 1);
      Key current_key = *(bucket->keys + key_offset);
      auto const found_or_empty_vote =
          g.ballot(current_key == EMPTY_KEY || insert_key == current_key);
      if (found_or_empty_vote) {
        found_or_empty = true;
        src_lane = __ffs(found_or_empty_vote) - 1;
        key_pos = (start_idx + tile_offset + src_lane) &
                  MAX_BUCKET_SIZE;
        if(rank == src_lane) {
          *(bucket->keys + key_pos) = insert_key;
          *(vectors + key_idx) = (bucket->vectors + key_pos);
          if (current_key == EMPTY_KEY) {
            d_sizes[bkt_idx]++;
          }
        }
        break;
      }
    }
    if (rank == 0) {
      if (key_pos == -1) {
        key_pos = bucket->min_pos;
        *(bucket->keys + key_pos) = insert_key;
        *(vectors + key_idx) = (bucket->vectors + key_pos);
      }
      /// Record storage offset. This will be used by write_kernel to map
      /// the input to the output data.

      if (src_offset != nullptr) {
        *(src_offset + key_idx) = key_idx;
      }
    }
  }
}
int main() {
  K *h_keys;
  K *d_keys;
  int *d_sizes;
  V **vectors;

  hipHostMalloc(&h_keys, KEY_NUM * sizeof(K));
  hipMalloc(&d_keys, KEY_NUM * sizeof(K));
  Bucket<K> *buckets;
  hipMallocManaged(&buckets, sizeof(Bucket<K>) * BUCKETS_NUM);
  for (int i = 0; i < BUCKETS_NUM; i++) {
    hipMalloc(&(buckets[i].keys), sizeof(K) * MAX_BUCKET_SIZE);
    hipMemset(buckets[i].keys, 0xFF, sizeof(K) * MAX_BUCKET_SIZE);
  }
  hipMalloc(&(d_sizes), sizeof(int) * BUCKETS_NUM);
  hipMemset(d_sizes, 0, sizeof(int) * BUCKETS_NUM);

  hipMalloc(&(vectors), sizeof(V *) * KEY_NUM);
  hipMemset(vectors, 0, sizeof(V *) * KEY_NUM);

  create_random_keys<K>(h_keys, KEY_NUM);
  hipMemcpy(d_keys, h_keys, KEY_NUM * sizeof(K), hipMemcpyHostToDevice);
  upsert_kernel<K><<<GRID_SIZE, BLOCK_SIZE>>>(d_keys, buckets, d_sizes, vectors,
                                              nullptr, N);
  hipDeviceSynchronize();

  create_random_keys<K>(h_keys, KEY_NUM);
  auto start_insert_or_assign = std::chrono::steady_clock::now();
  upsert_kernel<K><<<GRID_SIZE, BLOCK_SIZE>>>(d_keys, buckets, d_sizes, vectors,
                                              nullptr, N);
  hipDeviceSynchronize();
  auto end_insert_or_assign = std::chrono::steady_clock::now();

  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed1  : %s\n",
            hipGetErrorString(err));
    exit(-1);
  }

  for (int i = 0; i < BUCKETS_NUM; i++) {
    hipFree(buckets[i].keys);
  }
  hipFree(buckets);
  hipHostFree(h_keys);
  hipFree(d_keys);
  hipFree(d_sizes);
  hipFree(vectors);
  std::chrono::duration<double> diff_insert_or_assign =
      end_insert_or_assign - start_insert_or_assign;

  printf("[prepare] insert_or_assign=%.2fms\n",
         diff_insert_or_assign.count() * 1000);
  std::cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}
