
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <chrono>
#include <cstdio>
#include <cstdlib>

void usage(const char* filename) {
  printf("Sort the random key-value data set of the given length by key.\n");
  printf("Usage: %s <n>\n", filename);
}

constexpr int n = 1024 * 1024;

void random_vector(uint64_t* h_vec, size_t N) {
  static thrust::default_random_engine rng;
  static thrust::uniform_int_distribution<uint64_t> dist;

  for (size_t i = 0; i < N; i++) h_vec[i] = dist(rng);
}

using namespace thrust;

// TODO: Please refer to sorting examples:
// http://code.google.com/p/thrust/
// http://code.google.com/p/thrust/wiki/QuickStartGuide#Sorting

int main() {
  constexpr int TEST_TIMES = 1;
  const int printable_n = 16;

  // TODO: Generate random keys and values on host
  // host_vector<int> ...
  // generate( ...
  uint64_t* h_keys;
  uint64_t* h_vals;
  hipHostMalloc((void**)&h_keys, n * sizeof(uint64_t));
  hipHostMalloc((void**)&h_vals, n * sizeof(uint64_t));
  uint64_t* d_keys;
  uint64_t* d_vals;
  hipMalloc((void**)&d_keys, n * sizeof(uint64_t));
  hipMalloc((void**)&d_vals, n * sizeof(uint64_t));
  random_vector(h_keys, n);
  random_vector(h_vals, n);

  // Print out the input data if n is small.
  printf("Input data:\n");
  for (int i = 0; i < printable_n; i++)
    printf("(%d, %d)\n", h_keys[i], h_vals[i]);
  printf("\n");

  // TODO: Transfer data to the device.
  // device_vector<int> ...
  hipMemcpy(d_keys, h_keys, sizeof(uint64_t) * n, hipMemcpyHostToDevice);
  hipMemcpy(d_vals, h_vals, sizeof(uint64_t) * n, hipMemcpyHostToDevice);

  std::chrono::time_point<std::chrono::steady_clock> start_test;
  std::chrono::duration<double> diff_test;
  // TODO: Use sort_by_key or stable_sort_by_key to sort
  // pairs by key.
  // sort_by_key( ...
  start_test = std::chrono::steady_clock::now();
  thrust::device_ptr<uint64_t> d_keys_ptr(d_keys);
  thrust::device_ptr<uint64_t> d_vals_ptr(d_vals);
  thrust::sort_by_key(d_keys_ptr, d_keys_ptr + n, d_vals_ptr,
                      thrust::greater<uint64_t>());
  //   hipDeviceSynchronize();
  diff_test = std::chrono::steady_clock::now() - start_test;
  printf("[timing] sort d2h=%.2fms\n", diff_test.count() * 1000 / TEST_TIMES);

  // TODO: Transfer data back to host.
  hipMemcpy(h_keys, d_keys, sizeof(uint64_t) * n, hipMemcpyDeviceToHost);
  hipMemcpy(h_vals, d_vals, sizeof(uint64_t) * n, hipMemcpyDeviceToHost);

  // Print out the output data if n is small.
  printf("Output data:\n");
  for (int i = 0; i < printable_n; i++)
    printf("(%d, %d)\n", h_keys[i], h_vals[i]);
  printf("\n");

  hipHostFree(h_keys);
  hipHostFree(h_vals);
  hipFree(d_keys);
  hipFree(d_vals);

  return 0;
}
